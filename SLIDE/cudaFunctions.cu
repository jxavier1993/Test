#include "hip/hip_runtime.h"
#include "cudaFunctions.h"

__global__ void example_kernel(){
   printf("Hello from CUDA\n");
}

void call_example(){
   printf("able to call cuda function\n");
   example_kernel<<<1,1>>>();
   hipDeviceSynchronize();
}
